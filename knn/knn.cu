#include "hip/hip_runtime.h"
#include "knn.cuh"
#include <iostream>

using namespace std;

namespace knn {

void printCudaVersion() {
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    cout << "CUDA Runtime version: " << runtime_ver << '\n';

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << '\n';
}

void printCudaDevices() {
    int devicesCount;
    hipGetDeviceCount(&devicesCount);


    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex) {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        cout << "gpu[" << deviceIndex << "]: " << deviceProperties.name
             << " (" << deviceProperties.major << "." << deviceProperties.minor << ")\n";
    }
}


__global__ void knn_gpu(uint32_t V, uint32_t N, uint32_t Q, uint32_t K,
                        float* data, float* query, uint32_t* idx, float* dist) {

}



void knn_gpu_test(unsigned gpu_index,
                  uint32_t V, uint32_t N, uint32_t Q, uint32_t K,
                  float* data, float* query, uint32_t* idx, float* dist) {

    // get devices count
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    if (gpu_index >= devicesCount) {
        std::cout << "GPU index [" << gpu_index << "] out of range!" << '\n';
        return;
    }

    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, gpu_index);
    hipSetDevice(gpu_index);
    cout << "Using gpu[" << gpu_index << "]: " << deviceProperties.name
         << " (" << deviceProperties.major << "." << deviceProperties.minor << ")\n";

    // copy data to device
    float* d_data;
    hipMalloc((void**)&d_data, V * N * sizeof(float));
    hipMemcpy(d_data, data, V * N * sizeof(float), hipMemcpyHostToDevice);
    // copy query to device
    float* d_query;
    hipMalloc((void**)&d_query, V * Q * sizeof(float));
    hipMemcpy(d_query, query, V * Q * sizeof(float), hipMemcpyHostToDevice);
    // allocate indexes
    unsigned* d_idx;
    hipMalloc((void**)&d_idx, K * Q * sizeof(unsigned));
    // allocate distances
    float* d_dist;
    hipMalloc((void**)&d_dist, K * Q * sizeof(float));

    //knn_gpu<<<>>>(V, N, Q, K, data, query, idx, dist);

    // free
    hipFree(d_dist);
    hipFree(d_idx);
    hipFree(d_query);
    hipFree(d_data);


/*


        for (uint32_t q = 0; q < Q; q++) {
            float* local_query = query + V * q;
            float* local_dist = dist + K * q;
            uint32_t* local_idx = idx + K * q;
            // fill up distances with max float
            for (uint32_t k = 0; k < K; k++) {
                local_dist[k] = std::numeric_limits<float>::max();
            }
            for (uint32_t n = 0; n < N; n++) {
                const auto d = distance<V>(local_query, data + V * n);
                insert_in_order<K>(d, n, local_dist, local_idx);
            }
        }*/
}


}
